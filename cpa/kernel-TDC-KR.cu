#include "hip/hip_runtime.h"


#include <stdio.h>
#include "helpers.cuh"
#include "data.cuh"
#include <string>
#include <hip/hip_runtime.h>

#define SAMPLES 2000000
    // samples * wavelength  must to be less than 50,000,00
#define WAVELENGTH 600 // length of processed sampling points at a time
#define TOTAL  600	// total samplings in a power trace  // 280 for lattice 1024 for giii
#define KEYBYTES 16
#define KEYS 256
#define CHUNK 600	// number of chunk which must be loaded in to memory. chunk = wavelength *X ; x is an in
#define SEGMENTS 5
#define LIMIT (SAMPLES/SEGMENTS)	


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


typedef  unsigned char byte;
						//samples * CHUNK  must to be less than 4,000,000,000 this limit is from ram

__device__ byte hammingweight(byte M, byte R);
__device__ byte hamming(unsigned int *cipher, unsigned int i,unsigned int n,unsigned int key);
__global__ void maxCorelationkernel(double *corelation,double *wavestat,double *wavestat2,double *hammingstat, double *wavestatW2);
__global__ void wavestatkernel(unsigned char *wavedata, double *wavestat,double *wavestat2,byte *hammingArray, double *wavestatW2);
__global__ void hammingkernel(unsigned int *cipher, byte *hammingArray,double *hammingstat);

int main(int argc, char *argv[]){
//hipSetDevice(0);

//unsigned char dat;
float dat;
	
unsigned int i,j,k,temp;
//int l=atoi(argv[3]);
//char filename[80];
//strcpy(filename,"final");
//strcpy(filename,l+"");
//strcpy(filename,".txt");
FILE *file;
// empty the file. 
	file=fopen("all.txt","w");
	fprintf(file,"");
	fclose(file);
	
	double *MAXCorrelation=(double *)malloc(sizeof(double) * KEYS* KEYBYTES);
	isMemoryFull(MAXCorrelation);
	for (i=0;i<KEYS;i++){
		for(j=0;j<KEYBYTES;j++){
			MAXCorrelation[i*KEYBYTES+j]=0;
		}
	}
	
	//check args
	if(argc!=4){
		fprintf(stderr,"%s\n", "Not enough args. eg ./cpa wavedata.txt cipher.txt");
		exit(EXIT_FAILURE);
	}
	int ARRAY_CHUNKS = SAMPLES/LIMIT;
	
	//unsigned int *cipher[5];
	
	//for (i=0;i<5;i++){
	unsigned int *readCipher=(unsigned int *)malloc(sizeof(unsigned int)*(SAMPLES)*KEYBYTES);
	isMemoryFull(readCipher);

	//}

	unsigned int *cipher=(unsigned int *)malloc(sizeof(unsigned int)*(LIMIT)*KEYBYTES);
	isMemoryFull(cipher);

    unsigned char *readWaveData=(unsigned char *)malloc(1L*sizeof(byte) * SAMPLES*CHUNK);
	isMemoryFull(readWaveData);
	
	
//space for corelation
	double *corelation=(double *)malloc(sizeof(double) * KEYS * KEYBYTES);
	isMemoryFull(corelation);
	

	
//get cipher texts
			
	file=fopen(argv[2],"r");
	isFileOK(file);
	for(i=0; i<SAMPLES ;i++){
		int array_index = (i/LIMIT);
		//fprintf(stderr,"%d %f %d ",i, temp_1, array_index*LIMIT);
		//printf("%d %d %d ",i, array_index , array_index*LIMIT);
		for(j=0; j<KEYBYTES; j++){			
				fscanf(file,"%X",&readCipher[(i)*KEYBYTES+j]);
				//printf("%X",cipher[array_index][(i-(array_index*LIMIT))*KEYBYTES+j]);
		}
		//printf("%s","\n");
	}
	int numOfChunks=TOTAL/CHUNK;
	int l=0;
	for(l=0;l<numOfChunks;l++)
{	
		file=fopen(argv[1],"r");
		isFileOK(file);
		unsigned int rounds=TOTAL/WAVELENGTH;
		if(TOTAL%WAVELENGTH!=0) rounds++;
		int fileLength = strlen(argv[1]);
		char extention[4];
		strncpy(extention,argv[1]+fileLength-4,4) ;
		
		if(1){
	    fprintf(stderr,"%s\n",".data file is detected");
		//unsigned char  dat;
		
		//float dat;

/////////////////////	file read
	//fseek(file,sizeof(float)*CHUNK*l,SEEK_CUR);
		for(i=0; i<1*SAMPLES ;i++){
		   while(temp<CHUNK*l) {fread((void*)(&dat),sizeof(dat),1,file); temp++;}
		  
			temp=0;
			for(j=0; j<CHUNK; j++){
			fread((void*)(&dat),sizeof(dat),1,file);
                 	readWaveData[(i/1)*CHUNK+j]=(double)(dat);
					//if(dat>200)
					//  fprintf(stderr,"%f ",dat);
					}
			//fseek(file,sizeof(float)*(TOTAL-CHUNK),SEEK_CUR);
			while(temp<TOTAL-(CHUNK*(l+1))) {fread((void*)(&dat),sizeof(dat),1,file); temp++;}
			temp=0;
			//fprintf(stderr,"\n ");
				
			}
		}
		else{
		 long int dat;
			 fprintf(stderr,"%s\n",".txt file is detected");
			 for(i=0; i<SAMPLES ;i++){
	               	 for(j=0; j<WAVELENGTH; j++){
					fscanf(file,"%d",&dat); readWaveData[i*CHUNK+j]=(double)dat; 
					//printf("%ld ",dat);
							}
                        			}

			}
	
	//unsigned char *wavedata[5];
	//for(i=0; i<5 ;i++){	
	unsigned char *wavedata=(unsigned char *)malloc(sizeof(unsigned char) * (LIMIT)*  WAVELENGTH);
		isMemoryFull(wavedata);
	//}
	
			
	unsigned int insideRounds=CHUNK/WAVELENGTH;
	if(CHUNK%WAVELENGTH!=0) insideRounds++;
	for(k=0;k<insideRounds;k++) {   /// main loop
	
		//get wave data
	
	fprintf(stderr,"%s %d %d \n","calculating", l,k);
    
	
	//printf("%uc ",wavedata[0][0]);
	//Time
	
	//hipEvent_t start,stop;
	float elapsedtime;
	//hipEventCreate(&start);
	//hipEventRecord(start,0);
	//hipSetDevice(1);
	//cuda arrays and copying
	unsigned char *dev_wavedata;
	unsigned int *dev_cipher;
	
	double *dev_corelation,*dev_wavestat,*dev_wavestat2,*dev_hammingstat, *dev_wavestatW2;
	byte *dev_hammingArray;
	
	//gpuErrchk(hipMallocManaged((void**)&dev_wavedata, 5*sizeof(unsigned char*)));
	//gpuErrchk(hipMallocManaged((void**)&dev_cipher, 5*sizeof(unsigned int*)));
	//gpuErrchk(hipMallocManaged((void**)&dev_hammingArray, 5*sizeof(unsigned char*)));
	
	//for(i=0;i<5;i++){
		gpuErrchk(hipMallocManaged((void**)&dev_wavedata, 1L*(LIMIT)*WAVELENGTH*sizeof(unsigned char)));
	    hipGetLastError();
	
		gpuErrchk(hipMallocManaged((void**)&dev_cipher, 1L*(LIMIT)*KEYBYTES*sizeof(unsigned int)));
	    hipGetLastError();
		
		gpuErrchk(hipMallocManaged((void**)&dev_hammingArray, 1L*KEYS*KEYBYTES*(LIMIT)*sizeof(byte)));
	    hipGetLastError();
	
	//} // end for
	
	
	//gpuErrchk(hipMallocManaged((void**)&dev_wavedata1, 1L*(SAMPLES/2)*WAVELENGTH*sizeof(unsigned char)));
	//hipGetLastError();
	//gpuErrchk(hipMallocManaged((void**)&dev_cipher, 1L*(SAMPLES/2)*KEYBYTES*sizeof(unsigned int)));
	//hipGetLastError();
	//gpuErrchk(hipMallocManaged((void**)&dev_cipher1, 1L*(SAMPLES/2)*KEYBYTES*sizeof(unsigned int)));
	
	gpuErrchk(hipMallocManaged((void**)&dev_corelation, KEYS*KEYBYTES*sizeof(double)));
	hipGetLastError();
	gpuErrchk(hipMallocManaged((void**)&dev_wavestat, 1*WAVELENGTH*sizeof(double)));
	hipGetLastError();
	gpuErrchk(hipMallocManaged((void**)&dev_wavestatW2, 1*WAVELENGTH*sizeof(double)));
	hipGetLastError();
	gpuErrchk(hipMallocManaged((void**)&dev_wavestat2, 1L*KEYS*KEYBYTES*WAVELENGTH*sizeof(double)));
	hipGetLastError();
	gpuErrchk(hipMallocManaged((void**)&dev_hammingstat, 2*KEYS*KEYBYTES*sizeof(double)));
	hipGetLastError();
	
	dim3 grid(KEYBYTES/16,KEYS/16);
	dim3 block(16,16);
	
	dim3 block3d(16,16,4);
	dim3 grid3d(KEYBYTES/16,KEYS/16,WAVELENGTH/4);
	
	int array_index=0;
	
	hipMemset(&dev_wavestat, 0x00, 1*WAVELENGTH*sizeof(double));
	hipMemset(&dev_wavestatW2, 0x00, 1*WAVELENGTH*sizeof(double));
	hipMemset(&dev_wavestat2, 0x00, 1L*KEYS*KEYBYTES*WAVELENGTH*sizeof(double));
	hipMemset(&dev_hammingstat, 0x00, 2*KEYS*KEYBYTES*sizeof(double));
	
	for(int ir=0;ir<SEGMENTS;ir++){
	
	
	for(i=ir*LIMIT; i<(ir+1)*LIMIT ;i++){
		array_index = (i/LIMIT);
		for(j=0; j<KEYBYTES; j++){
			unsigned int temp=readCipher[(i)*KEYBYTES+j];
			cipher[(i-(array_index*LIMIT))*KEYBYTES+j] = temp;
		}
		for(j=0; j<WAVELENGTH; j++){
			double value1= readWaveData[(i)*CHUNK+j+k*WAVELENGTH];
			wavedata[(i-array_index*LIMIT)*WAVELENGTH+j] =(unsigned char)value1;
		
	
		}
	}
		
		gpuErrchk(hipMemcpy(dev_wavedata,wavedata,1L*(LIMIT)*WAVELENGTH*sizeof(unsigned char),hipMemcpyHostToDevice));
		hipGetLastError();
		
		gpuErrchk(hipMemcpy(dev_cipher,cipher, 1L*(LIMIT)*KEYBYTES*sizeof(unsigned int),hipMemcpyHostToDevice));
		hipGetLastError();
	
		//findhamming
		hammingkernel<<<grid,block>>>(dev_cipher,dev_hammingArray,dev_hammingstat);
		hipDeviceSynchronize();
		hipGetLastError();

	
		//find wave stats
		wavestatkernel<<<grid3d,block3d>>>(dev_wavedata,dev_wavestat,dev_wavestat2,dev_hammingArray,dev_wavestatW2);
		hipDeviceSynchronize();
		hipGetLastError();
	
		
		}  // end for
	
	//gpuErrchk(hipMallocManaged((void**)&dev_Numerator, 1L*KEYS*KEYBYTES*WAVELENGTH*sizeof(double)));
	//gpuErrchk(hipMallocManaged((void**)&dev_Denominator, 1L*KEYS*KEYBYTES*WAVELENGTH*sizeof(double)));
	
	//deploy double 
	maxCorelationkernel<<<grid,block>>>(dev_corelation,dev_wavestat,dev_wavestat2,dev_hammingstat,dev_wavestatW2);
	hipDeviceSynchronize();
	hipGetLastError();


	//copy back
	hipMemcpy(corelation, dev_corelation, KEYS*KEYBYTES*sizeof(double),hipMemcpyDeviceToHost);
	
	
	//for (i=0;i<KEYBYTES;i++){
	//
	//	for (j=0;j<KEYS;j++){
	//		double max_correlation =0.0;
	//		
	//		for (k=0;k<WAVELENGTH;k++){
	//			
	//		double N = Numerator[i*KEYS*WAVELENGTH+ j* WAVELENGTH+ k];
	//		double D = Denominator[i*KEYS*WAVELENGTH+ j* WAVELENGTH+ k];
	//		
	//		double corr = fabs(N/D);
	//		if(corr > max_correlation)
	//			max_correlation   =  corr;		
	//			
	//		}
	//		//corelation[i*KEYBYTES+j] = max_correlation; 
	//					
	//	}
	//
	//}
	
	
	//for(i=0;i<5;i++){
		hipFree(dev_wavedata);
		hipFree(dev_cipher);
		hipFree(dev_hammingArray);
	//}
		
	//hipFree(dev_wavedata);
	//hipFree(dev_cipher);
	hipFree(dev_corelation);
	hipFree(dev_wavestat);
	hipFree(dev_wavestat2);
	hipFree(dev_wavestatW2);
	
	hipFree(dev_hammingstat);
	//hipFree(dev_hammingArray);
	//hipFree(dev_Denominator);
	//hipFree(dev_Numerator);
	
	
	
	
	
	//Time
	//hipEventCreate(&stop);
	//hipEventRecord(stop,0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&elapsedtime,start,stop);
	//fprintf(stderr,"Time spent for CUDA operation : %.10f\n",elapsedtime/(float)1000);

	//char finishtime[30];
	//time_t stopRound=time(NULL);
	//struct tm *loctime;
	//loctime = localtime (&stopRound);
	//strftime (finishtime, 30, "%Y-%m-%d %H:%M:%S", loctime);
	
	file=fopen("all.txt","a");
	fprintf(file,"\n");
	//fprintf(file,"%s \n",finishtime);
	fprintf(file,"%d,  pk0,   pk1,   pk2,   pk3,   pk4,   pk5,   pk6,   pk7,   pk8,   pk9,  pk10,  pk11,  pk12,  pk13,  pk14,  pk15, \n",SAMPLES);
	for (i=0;i<KEYS;i++){
	fprintf(file,"0x%02X,",i);
		for(j=0;j<KEYBYTES;j++){
			fprintf(file,"%.5f,",i,corelation[i*KEYBYTES+j]);
			double value1 =MAXCorrelation[i*KEYBYTES+j];
			double value2 = corelation[i*KEYBYTES+j];
			
			if(value1<value2) 
				MAXCorrelation[i*KEYBYTES+j] = value2;			
		}
		fprintf(file,"\n");
	}
	fprintf(file,"\n");
	fclose(file);

	
	} // inside loop
	

}
free(readWaveData);

//free(wavedata);
//free(wavedata1);

double finalCorrelations[KEYS][KEYBYTES];
double n=0; int positions[KEYS][KEYBYTES];

	char finishtime[30];
 	time_t stop=time(NULL);
	//finishtime= ctime(&stop);  
	struct tm *loctime;
	loctime = localtime (&stop);
	strftime (finishtime, 30, "%Y-%m-%d %H:%M:%S", loctime);
	
	char krFilePath[30] = "./kr_files/";
	
	char krFileName[20];
	sprintf(krFileName, "%d", SAMPLES);
	
	strcat(krFileName, ".txt");
	
	strcat(krFilePath, krFileName);
	printf("%s ", krFilePath);
	
	// write the csv file
	file=fopen(krFilePath,"w");
	//fprintf(file,"\n");
	//fprintf(file,"%s \n",finishtime);
	//fprintf(file,"%d,  pk0,  pk1,  pk2,  pk3,  pk4,  pk5,  pk6,  pk7,  pk8,  pk9, pk10, pk11, pk12, pk13, pk14, pk15, \n",SAMPLES);
	for (i=0;i<KEYS;i++){
	//fprintf(file,"0x%02X,",i);
		for(j=0;j<KEYBYTES;j++){
			if(j==KEYBYTES-1)
				fprintf(file,"%.5f",MAXCorrelation[i*KEYBYTES+j]);
			else
				fprintf(file,"%.5f,",MAXCorrelation[i*KEYBYTES+j]);
			finalCorrelations[i][j]=MAXCorrelation[i*KEYBYTES+j];
			}
		fprintf(file,"\n");
	}
////////////////////////////// sort the results /// 
	int p=0;
	for(j=0;j<KEYBYTES;j++){
		for(i=0;i<KEYS;i++) positions[i][j] =i;
		for (p=0;p<255;p++){
		
		for (i=0;i<KEYS-p-1;i++){
					if(finalCorrelations[i][j]<finalCorrelations[i+1][j]) { 
						n=finalCorrelations[i][j];
						finalCorrelations[i][j]=finalCorrelations[i+1][j];
						finalCorrelations[i+1][j]=n; 
					
						n=positions[i][j];
						positions[i][j]=positions[i+1][j];
						positions[i+1][j]=n; 
						}
					
					}
			}
	}
	
	for(j=0;j<KEYBYTES;j++){
	printf("    |%d|\t",j);
	}
	printf("\n");
	
	for (i=0;i<5;i++){
	for(j=0;j<KEYBYTES;j++){
		printf("   %02x  \t",positions[i][j]);
		}
		printf("\n");
	for(j=0;j<KEYBYTES;j++){
		printf("%.5f  ",finalCorrelations[i][j]);
		}
		printf("\n\n");
	}
		
	
	
	return 0;
}


__device__ byte hammingweight(byte M, byte R){
	byte H=M^R;
	// Count the number of set bits
	byte dist=0;
	while(H){
		dist++; 
		H &= H - 1;
}
	

	return dist;
}

__device__ byte hamming(unsigned int *cipher, unsigned int i,unsigned int n,unsigned int key) { //n is byteno  i is the sample
      byte st10 = (byte)cipher[i*KEYBYTES+inv_shift[n]];
      byte st9 = (byte)inv_sbox[cipher[i*KEYBYTES+n]  ^ key] ;
      byte st19 = (byte)(inv_sbox[cipher[i*KEYBYTES+inv_shift[n] ]  ^ key]) ;

//   byte st9 = (byte)((tbox[cipher[i*KEYBYTES+n]  ^ key]>>24));
//   byte st9 = (byte)(inv_sbox[cipher[i*KEYBYTES+n]  ^ key ]);
//  byte st10 = (byte)(cipher[i*KEYBYTES+n] ^key );

//	byte dist=0;
//if((st9 ^ st10) & 0x0FF)
//idf((st9 ^ st10) & 0x0)
//		 dist= 1;
//	else
//		 dist= 0;  

   byte dist = hammingweight(st9,st10);
 

// byte dist = hammingweight(st9&0xf0,st10&0xf0);	  
//   byte dist = hammingweight(st9, 0);
// byte dist = hammingweight(st19 ,0);

// byte dist = hammingweight(st9, 0);
//

	return dist;
}


__global__ void maxCorelationkernel(double *corelation,double *wavestat,double *wavestat2,double *hammingstat, double *wavestatW2){ 
	
	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;

	if (keybyte<KEYBYTES && keyguess<KEYS ){ 
		
		double sigmaH,sigmaH2,sigmaW=0,sigmaW2=0,sigmaWH=0;	
		sigmaH=hammingstat[KEYBYTES*keyguess+keybyte];    					// already log 10
		sigmaH2=hammingstat[KEYS*KEYBYTES+KEYBYTES*keyguess+keybyte];		// already log 10
		double temp_corelation=0;;
		double corelationmax=0;
		unsigned int j;
		double logSample=  __log10f(SAMPLES);
		
		for(j=0;j<WAVELENGTH;j++){
			
			sigmaWH=wavestat2[j*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte]; // already log 10
			sigmaW=wavestat[j];                                               // already log 10 
			sigmaW2=wavestatW2[j];                                   // already log10
	
						
			double denominator=sqrt((SAMPLES)*sigmaW2 - sigmaW*sigmaW)*sqrt((SAMPLES)*sigmaH2 - sigmaH*sigmaH);
			double numerator  =(SAMPLES)*sigmaWH - sigmaW*sigmaH;
			
			//Numerator[keybyte*KEYS*WAVELENGTH+ keyguess* WAVELENGTH+ j] = numerator;
			//Denominator[keybyte*KEYS*WAVELENGTH+ keyguess* WAVELENGTH+ j] = denominator;
			
			temp_corelation= fabs((double)numerator/(double)denominator);
			
			if(temp_corelation>corelationmax){
				corelationmax=temp_corelation; 
			}
		}

		corelation[keyguess*KEYBYTES+keybyte]=corelationmax;
	}
	return;
}

__global__ void wavestatkernel(unsigned char *wavedata, double *wavestat,double *wavestat2,byte *hammingArray, double *wavestatW2){

	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;
	int wave=blockDim.z*blockIdx.z+threadIdx.z;

	if (keyguess<KEYS && keybyte<KEYBYTES && wave<WAVELENGTH ){
		unsigned int i=0, j=0;
		double sigmaWH=0;
		//for(j=0;j<5;j++){
			for(i=0;i<LIMIT;i++){
				sigmaWH+=wavedata[i*WAVELENGTH+wave]*(double)hammingArray[i*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte];	
			}
		//}
		
		
		wavestat2[wave*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte ]  = wavestat2[wave*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte ] + (sigmaWH);
	}

	if (keyguess==0 && keybyte==0 && wave<WAVELENGTH ){
		unsigned long i=0, j=0;
		double sigmaW=0,sigmaW2=0,W=0, temp=0;
		
		//for(j=0;j<5;j++){
		
			for(i=0;i<LIMIT;i++){
				W=wavedata[i*WAVELENGTH+wave];
				sigmaW+=W;
				temp =(W*W);
				sigmaW2 = sigmaW2 + temp;// temp;
			}
		//}

		wavestat[wave]  =wavestat[wave]+(sigmaW);
		wavestatW2[wave]=wavestatW2[wave]+(sigmaW2);
		
	}
	return;
}

__global__ void hammingkernel(unsigned int *cipher,byte *hammingArray,double *hammingstat){
	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;

	if (keybyte<KEYBYTES && keyguess<KEYS ){
		double sigmaH=0,sigmaH2=0, temp=0;
		byte H;
		unsigned int i=0, j=0;
		//for(j=0;j<5;j++){
			for(i=0;i<LIMIT;i++){
				H=hamming(cipher,i,keybyte,keyguess);
				hammingArray[(i)*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte]= (H);
				sigmaH+=(double)H;
				temp   =(H*H);
				sigmaH2=sigmaH2 + temp;
			}
		//}
		
		hammingstat[KEYBYTES*keyguess+keybyte]  = hammingstat[KEYBYTES*keyguess+keybyte]+ (sigmaH);
		hammingstat[KEYS*KEYBYTES+KEYBYTES*keyguess+keybyte]=hammingstat[KEYS*KEYBYTES+KEYBYTES*keyguess+keybyte]+sigmaH2;
	}
	return;
}

